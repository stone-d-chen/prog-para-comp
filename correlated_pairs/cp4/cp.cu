/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <hip/hip_runtime.h>

typedef float f32;

__global__ void kernel(int ny, int nx, const float *data, float *result)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if(i >= ny || j >= ny) return;

    f32 v = 0;
    for(int k = 0; k < nx; ++k)
    {
        v += data[nx*i + k] * data[nx * j + k];
    }

    result[ny * i + j] = v;
}

int divup(int n, int factor) {
    int Result = (n + factor - 1) / factor;
    return(Result);
}

void correlate(int ny, int nx, const float *data, float *result)
{
    f32 *NormData = (f32 *)malloc(ny*nx*sizeof(f32));

    for(int y = 0; y < ny; ++y)
    {
        f32 sum = 0;
        for(int x = 0; x < nx; ++x)
        {
            f32 val = data[nx * y + x];
            NormData[nx*y + x] = val;
            sum += val;
        }
        int mean = sum/nx;

        f32 SumSq = 0;
        for(int x = 0; x < nx; ++x)
        {
            NormData[nx*y + x] -= mean;
            SumSq += NormData[nx*y + x] * NormData[nx*y + x];
        }

        f32 InvStd = 1/sqrtf(SumSq);

        for(int x = 0; x < nx; ++x)
        {
            NormData[nx*y + x] *= InvStd;
        }
    }

    f32 Result = 0;
    for(int j = 0; j < ny; ++j)
    {
        for(int i = j; i < ny; ++i)
        {
            for(int k = 0; k < nx; ++k)
            {
                f32 x = NormData[nx*i + k];
                f32 y = NormData[nx*j + k];
                Result += x * y;
            }
            result[ny*j + i] = Result;
        }
    }

    dim3 dimBlock(16,16);
    dim3 dimGrid(divup(nx, 16), divup(ny, 16));

    f32 *dataGPU;
    f32 *resultGPU;

   hipMalloc((void**)&dataGPU, sizeof(f32) * ny * nx);
   hipMalloc((void**)&resultGPU, sizeof(f32) * ny * ny);

   hipMemcpy(dataGPU, data, sizeof(f32) * ny * nx, hipMemcpyHostToDevice);

   kernel <<< dimGrid, dimBlock >>>(nx, ny, dataGPU, resultGPU);

   hipDeviceSynchronize();

   hipMemcpy(result, resultGPU, sizeof(f32) * ny * ny, hipMemcpyDeviceToHost);

   hipFree(dataGPU);
   hipFree(resultGPU);

}
